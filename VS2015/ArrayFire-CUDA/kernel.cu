#include "hip/hip_runtime.h"
#include <arrayfire.h>

using namespace af;

#include <stdio.h>

#define divup(x,y) (x%y) ? ((x+y-1)/y) : (x/y)

#define CUDA(call) do {                                         \
        hipError_t _e = (call);                                \
        if (_e == hipSuccess) break;                           \
        fprintf(stderr, __FILE__":%d: cuda error: %s (%d)\n",   \
                __LINE__, hipGetErrorString(_e), _e);          \
        exit(-1);                                               \
        } while (0)


// generate millions of random elements
static int elements = 1e5;

static int reduction_cpu(const int *input)
{
    int sum = 0;
    for (int i = 0; i < elements; ++i) {
        sum += input[i];
    }
    return sum;
}

static int reduction_af(const array input)
{
    return sum<int>(input);
}

__global__ void reduction_kernel(int *d_odata, const int *d_idata, const int n)
{
    extern __shared__ int smem[];
    smem[threadIdx.x] = 0;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        smem[threadIdx.x] = d_idata[idx];
    __syncthreads();
    for (int c = blockDim.x / 2; c > 0; c /= 2)
    {
        if (threadIdx.x < c)
            smem[threadIdx.x] += smem[threadIdx.x + c];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        d_odata[blockIdx.x] = smem[0];
}

static int reduction_cuda(const int *d_idata)
{
    // TODO determine numBlocks and numThreads
    int numThreads = 512;
    int numBlocks = divup(elements, numThreads);

    // allocate device memory and data
    int *d_odata = NULL;
    CUDA(hipMalloc((void **)&d_odata, numBlocks*sizeof(int)));

    // Call your reduce kernel(s) with the right parameters
    // INPUT:       d_idata
    // OUTPUT:      d_odata
    // (1) reduce across all elements
    reduction_kernel << <numBlocks, numThreads, sizeof(int) * numThreads >> >(d_odata, d_idata, elements);
    CUDA(hipDeviceSynchronize());
    // (2) reduce across all blocks
    int gpu_result = 0;
    size_t block_bytes = numBlocks * sizeof(int);
    int *h_blocks = (int *)malloc(block_bytes);
    CUDA(hipMemcpy(h_blocks, d_odata, block_bytes, hipMemcpyDeviceToHost));
    for (int i = 0; i < numBlocks; ++i)
        gpu_result += h_blocks[i];

    // cleanup
    CUDA(hipFree(d_odata));
    free(h_blocks);

    return gpu_result;
}

int main(int argc, char* argv[])
{
    try {
        // perform timings and calculate error from reference PI
        info();

        // Create random array on device
        array input = (randu(elements) * 100).as(s32);

        // Do redution using ArrayFire
        int sum_af = reduction_af(input);
        printf("arrayfire result = %d\n", sum_af);

        // Copy device array to host and get a pointer
        std::vector<int> h_input(input.elements());
        input.host(&h_input.front());
        // Do reduction using CPU
        int sum_cpu = reduction_cpu(&h_input.front());
        printf("cpu result       = %d\n", sum_cpu);

        // Get device pointer. No copy
        int *d_input = input.device<int>();
        // Do reduction on CUDA
        int sum_cuda = reduction_cuda(d_input);
        printf("cuda result      = %d\n", sum_cuda);
    }
    catch (af::exception& e) {
        fprintf(stderr, "%s\n", e.what());
        throw;
    }

#ifdef WIN32 // pause in Windows
    if (!(argc == 2 && argv[1][0] == '-')) {
        printf("hit [enter]...");
        getchar();
    }
#endif
    return 0;
}